/*
 * Author: Guillaume HELBECQUE (Université du Luxembourg)
 * Date: 10/10/2024
 *
 * Description:
 * This program solves the N-Queens problem using a sequential Depth-First tree-Search
 * (DFS) algorithm. It serves as a basis for task-parallel implementations.
 */

#include <iostream>
#include <cstring>
#include <utility>
#include <vector>
#include <chrono>
#include <stack>
#include <hip/hip_runtime.h>

// Static functions for inequalities
static bool no_same_column(const std::vector<int> &board, int row, int col)
{
  for (int i = 0; i < row; ++i)
  {
    if (board[i] == col)
    {
      return false;
    }
  }
  return true;
}

static bool no_same_diagonal(const std::vector<int> &board, int row, int col)
{
  for (int i = 0; i < row; ++i)
  {
    if (board[i] == col - row + i || board[i] == col + row - i)
    {
      return false;
    }
  }
  return true;
}

static bool no_column_zero_if_even_row(const std::vector<int> &board, int row, int col)
{
  return !(row % 2 == 0 && col == 0);
}

__global__ void propagate_domains_kernel(bool *domain, int *board, int depth, size_t N)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x; // Map thread to row
    int col = blockIdx.y * blockDim.y + threadIdx.y; // Map thread to column

    // Ensure the thread indices are within bounds
    if (row >= depth && row < N && col < N)
    {
        // Iterate over all previously placed queens
        for (int placed_row = 0; placed_row < depth; ++placed_row)
        {
            int placed_col = board[placed_row]; // Column of the queen in `placed_row`

            // Remove column conflicts
            if (col == placed_col)
                domain[row * N + col] = false;

            // Remove diagonal conflicts
            if (col == placed_col - (row - placed_row) || col == placed_col + (row - placed_row))
                domain[row * N + col] = false;
        }
    }
}

// N-Queens node
struct Node
{
  int depth;                            // Depth in the tree
  std::vector<int> board;               // Board configuration (permutation)
  std::vector<std::vector<bool> > domain; // Domain for each row

  Node(size_t N) : depth(0), board(N), domain(N, std::vector<bool>(N, true))
  {
    for (int i = 0; i < N; i++)
    {
      board[i] = i; // Initialize board with default column indices
    }
  }
  Node(const Node &) = default;
  Node(Node &&) = default;
  Node() = default;
};

// Apply arbitrary inequalities
bool check_inequalities(const std::vector<int> &board, int row, int col,
                        const std::vector<bool (*)(const std::vector<int> &, int, int)> &inequalities)
{
  for (const auto &inequality : inequalities)
  {
    if (!inequality(board, row, col))
    {
      return false;
    }
  }
  return true;
}
bool propagate_domains(Node &node, size_t N)
{
    // Allocate device memory for domain and board
    bool *d_domain;
    int *d_board;
    hipMalloc(&d_domain, N * N * sizeof(bool));
    hipMalloc(&d_board, N * sizeof(int));

    // Copy data to device
    hipMemcpy(d_domain, &node.domain[0][0], N * N * sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(d_board, node.board.data(), N * sizeof(int), hipMemcpyHostToDevice);

    // Define block and grid sizes dynamically based on N
    int blockDimX = 16; // Threads per block in the X dimension (columns)
    int blockDimY = 16; // Threads per block in the Y dimension (rows)

    dim3 blockSize(blockDimX, blockDimY);
    dim3 gridSize((N + blockDimX - 1) / blockDimX, (N + blockDimY - 1) / blockDimY);

    // Fixed number of iterations for domain propagation
    const int max_iterations = 10;
    for (int iter = 0; iter < max_iterations; ++iter)
    {
        propagate_domains_kernel<<<gridSize, blockSize>>>(d_domain, d_board, node.depth, N);
        hipDeviceSynchronize(); // Ensure kernel execution is completed before next iteration
    }

    // Copy the updated domain back to the host
    hipMemcpy(&node.domain[0][0], d_domain, N * N * sizeof(bool), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_domain);
    hipFree(d_board);

    // Check if any row has an empty domain
    for (int row = node.depth; row < N; ++row)
    {
        bool hasValidValue = false;
        for (int col = 0; col < N; ++col)
        {
            if (node.domain[row][col])
            {
                hasValidValue = true;
                break;
            }
        }
        if (!hasValidValue)
        {
            return false; // Dead-end: No valid placements possible for this node
        }
    }

    return true; // Valid node
}

// Evaluate and branch function with fixpoint domain propagation
void evaluate_and_branch(const Node &parent, std::stack<Node> &pool, size_t &tree_loc, size_t &num_sol,
                         const std::vector<bool (*)(const std::vector<int> &, int, int)> &inequalities)
{
    int depth = parent.depth;
    int N = parent.board.size();

    // If the node is a leaf, count it as a solution
    if (depth == N)
    {
        num_sol++;
        return;
    }

    // Iterate over the domain of the current row
    for (int col = 0; col < N; ++col)
    {
        if (parent.domain[depth][col]) // Check if column is in the domain
        {
            Node child(parent);
            child.board[depth] = col; // Place the queen
            child.depth++;

            // Reduce domain for the child and propagate fixpoint
            if (propagate_domains(child, N))
            {
                pool.push(std::move(child));
                tree_loc++;
            }
        }
    }
}

int main(int argc, char **argv)
{
  // helper
  if (argc != 2)
  {
    std::cout << "usage: " << argv[0] << " <number of queens> " << std::endl;
    exit(1);
  }

  std::vector<bool (*)(const std::vector<int> &, int, int)> inequalities;
  inequalities.push_back(no_same_column);
  inequalities.push_back(no_same_diagonal);
  // inequalities.push_back(no_column_zero_if_even_row);

  // problem size (number of queens)
  size_t N = std::stoll(argv[1]);
  std::cout << "Solving " << N << "-Queens problem\n"
            << std::endl;

  // initialization of the root node (the board configuration where no queen is placed)
  Node root(N);

  // initialization of the pool of nodes (stack -> DFS exploration order)
  std::stack<Node> pool;
  pool.push(std::move(root));

  // statistics to check correctness (number of nodes explored and number of solutions found)
  size_t exploredTree = 0;
  size_t exploredSol = 0;

  // beginning of the Depth-First tree-Search
  auto start = std::chrono::steady_clock::now();

  while (pool.size() != 0)
  {
    // get a node from the pool
    Node currentNode(std::move(pool.top()));
    pool.pop();

    // check the board configuration of the node and branch it if it is valid.
    evaluate_and_branch(currentNode, pool, exploredTree, exploredSol, inequalities);
  }

  auto end = std::chrono::steady_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);

  // outputs
  std::cout << "Time taken: " << duration.count() << " milliseconds" << std::endl;
  std::cout << "Total solutions: " << exploredSol << std::endl;
  std::cout << "Size of the explored tree: " << exploredTree << std::endl;

  return 0;
}
