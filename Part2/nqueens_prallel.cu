/*
 * Author: Guillaume HELBECQUE (Université du Luxembourg)
 * Date: 10/10/2024
 *
 * Description:
 * This program solves the N-Queens problem using a sequential Depth-First tree-Search
 * (DFS) algorithm. It serves as a basis for task-parallel implementations.
 */

#include <iostream>
#include <cstring>
#include <utility>
#include <vector>
#include <chrono>
#include <stack>
#include <hip/hip_runtime.h>

// Static functions for inequalities
static bool no_same_column(const std::vector<int> &board, int row, int col)
{
  for (int i = 0; i < row; ++i)
  {
    if (board[i] == col)
    {
      return false;
    }
  }
  return true;
}

static bool no_same_diagonal(const std::vector<int> &board, int row, int col)
{
  for (int i = 0; i < row; ++i)
  {
    if (board[i] == col - row + i || board[i] == col + row - i)
    {
      return false;
    }
  }
  return true;
}

static bool no_column_zero_if_even_row(const std::vector<int> &board, int row, int col)
{
  return !(row % 2 == 0 && col == 0);
}

__global__ void parallel_propagate(bool *domain, const int *board, int depth, size_t N)
{
  int row = blockIdx.x * blockDim.x + threadIdx.x; // Thread index for row
  int col = blockIdx.y * blockDim.y + threadIdx.y; // Thread index for column

  if (row >= depth && row < N && col < N) // Ensure within bounds
  {
    for (int placed_row = 0; placed_row < depth; ++placed_row)
    {
      int placed_col = board[placed_row]; // Queen already placed at this column

      // Remove column conflicts
      if (col == placed_col)
      {
        domain[row * N + col] = false;
      }

      // Remove diagonal conflicts
      if (col == placed_col - (row - placed_row) || col == placed_col + (row - placed_row))
      {
        domain[row * N + col] = false;
      }
    }
  }
}

// N-Queens node
struct Node
{
  int depth;                             // Depth in the tree
  std::vector<int> board;                // Board configuration (permutation)
  std::vector<std::vector<bool>> domain; // Domain for each row

  Node(size_t N) : depth(0), board(N), domain(N, std::vector<bool>(N, true))
  {
    for (int i = 0; i < N; i++)
    {
      board[i] = i; // Initialize board with default column indices
    }
  }
  Node(const Node &) = default;
  Node(Node &&) = default;
  Node() = default;
};

// Apply arbitrary inequalities
bool check_inequalities(const std::vector<int> &board, int row, int col,
                        const std::vector<bool (*)(const std::vector<int> &, int, int)> &inequalities)
{
  for (const auto &inequality : inequalities)
  {
    if (!inequality(board, row, col))
    {
      return false;
    }
  }
  return true;
}
bool propagate_domains(Node &node, size_t N)
{
  // Allocate device memory
  bool *d_domain;
  int *d_board;
  hipMalloc(&d_domain, N * N * sizeof(bool));
  hipMalloc(&d_board, N * sizeof(int));

  // Copy initial data to device
  hipMemcpy(d_domain, &node.domain[0][0], N * N * sizeof(bool), hipMemcpyHostToDevice);
  hipMemcpy(d_board, node.board.data(), N * sizeof(int), hipMemcpyHostToDevice);

  // Define grid and block sizes
  int blockDimX = 16;
  int blockDimY = 16;
  dim3 blockSize(blockDimX, blockDimY);
  dim3 gridSize((N + blockDimX - 1) / blockDimX, (N + blockDimY - 1) / blockDimY);

  // Fixpoint loop
  const int max_iterations = 10; // Set based on expected convergence
  for (int iter = 0; iter < max_iterations; ++iter)
  {
    parallel_propagate<<<gridSize, blockSize>>>(d_domain, d_board, node.depth, N);
    hipDeviceSynchronize();
  }

  // Copy the final domain back to the host
  hipMemcpy(&node.domain[0][0], d_domain, N * N * sizeof(bool), hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_domain);
  hipFree(d_board);

  // Check if any row has an empty domain (indicating a dead-end)
  for (int row = node.depth; row < N; ++row)
  {
    bool hasValidValue = false;
    for (int col = 0; col < N; ++col)
    {
      if (node.domain[row][col])
      {
        hasValidValue = true;
        break;
      }
    }
    if (!hasValidValue)
    {
      return false; // Dead-end: No valid placements
    }
  }

  return true; // Valid node
}

// Evaluate and branch function with fixpoint domain propagation
void evaluate_and_branch(const Node &parent, std::stack<Node> &pool, size_t &tree_loc, size_t &num_sol,
                         const std::vector<bool (*)(const std::vector<int> &, int, int)> &inequalities)
{
  int depth = parent.depth;
  int N = parent.board.size();

  // If the node is a leaf, count it as a solution
  if (depth == N)
  {
    num_sol++;
    return;
  }

  // Iterate over the domain of the current row
  for (int col = 0; col < N; ++col)
  {
    if (parent.domain[depth][col]) // Check if column is in the domain
    {
      Node child(parent);
      child.board[depth] = col; // Place the queen
      child.depth++;

      // Reduce domain for the child and propagate fixpoint
      if (propagate_domains(child, N))
      {
        pool.push(std::move(child));
        tree_loc++;
      }
    }
  }
}

int main(int argc, char **argv)
{
  // helper
  if (argc != 2)
  {
    std::cout << "usage: " << argv[0] << " <number of queens> " << std::endl;
    exit(1);
  }

  std::vector<bool (*)(const std::vector<int> &, int, int)> inequalities;
  inequalities.push_back(no_same_column);
  inequalities.push_back(no_same_diagonal);
  // inequalities.push_back(no_column_zero_if_even_row);

  // problem size (number of queens)
  size_t N = std::stoll(argv[1]);
  std::cout << "Solving " << N << "-Queens problem\n"
            << std::endl;

  // initialization of the root node (the board configuration where no queen is placed)
  Node root(N);

  // initialization of the pool of nodes (stack -> DFS exploration order)
  std::stack<Node> pool;
  pool.push(std::move(root));

  // statistics to check correctness (number of nodes explored and number of solutions found)
  size_t exploredTree = 0;
  size_t exploredSol = 0;

  // beginning of the Depth-First tree-Search
  auto start = std::chrono::steady_clock::now();

  while (pool.size() != 0)
  {
    // get a node from the pool
    Node currentNode(std::move(pool.top()));
    pool.pop();

    // check the board configuration of the node and branch it if it is valid.
    evaluate_and_branch(currentNode, pool, exploredTree, exploredSol, inequalities);
  }

  auto end = std::chrono::steady_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);

  // outputs
  std::cout << "Time taken: " << duration.count() << " milliseconds" << std::endl;
  std::cout << "Total solutions: " << exploredSol << std::endl;
  std::cout << "Size of the explored tree: " << exploredTree << std::endl;

  return 0;
}
